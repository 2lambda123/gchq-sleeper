#include "hip/hip_runtime.h"

#include "cudf_compact/ranges.hpp"

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <spdlog/spdlog.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include "cudf_compact/filters.hpp"

template<typename UnaryFunction>
inline __device__ auto make_counting_transform_iterator(cudf::size_type start, UnaryFunction f)
{
    return thrust::make_transform_iterator(thrust::make_counting_iterator(start), f);
}

struct row_total_size
{
    page_info const *cum_pages;
    int const *key_offsets;
    size_t num_keys;

    __device__ inline page_info operator()(page_info const &i)
    {
        // sum sizes for each input column at this row
        size_t sum = 0;
        for (int idx = 0; idx < num_keys; idx++) {
            auto const start = key_offsets[idx];
            auto const end = key_offsets[idx + 1];
            auto iter = make_counting_transform_iterator(0, [&] __device__(int i) { return cum_pages[i].row_count; });
            auto const page_index = thrust::lower_bound(thrust::seq, iter + start, iter + end, i.row_count) - iter;
            sum += cum_pages[page_index].size_bytes;
        }
        return { i.file_idx, i.rg_idx, i.col_idx, i.page_idx, i.schema_idx, i.global_col_idx, i.row_count, sum };
    }
};

struct page_info_by_index
{
    page_info *data;

    __device__ inline page_info operator()(int index) { return data[index]; }
};

std::deque<scalar_pair> getRanges(std::vector<page_info> const &pages,
  cudf::size_type sort_col,
  parquet::format::Type::type col_type,
  parquet::format::ConvertedType::type conv_type,
  size_t chunk_read_limit,
  std::vector<std::vector<parquet::format::ColumnIndex>> const &indexes_per_file)
{
    auto stream = rmm::cuda_stream_default;

    // create page keys and copy to device
    std::vector<int> h_page_keys(pages.size());
    std::transform(
      pages.begin(), pages.end(), h_page_keys.begin(), [](auto const &page) { return page.global_col_idx; });

    rmm::device_uvector<int> page_keys(pages.size(), stream);
    hipMemcpyAsync(page_keys.data(), h_page_keys.data(), sizeof(int) * h_page_keys.size(), hipMemcpyDefault, stream);

    rmm::device_uvector<int> page_index(page_keys.size(), stream);
    thrust::sequence(thrust::device, page_index.begin(), page_index.end());

    thrust::stable_sort_by_key(
      thrust::device, page_keys.begin(), page_keys.end(), page_index.begin(), thrust::less<int>());

    // copy pages vector to device
    rmm::device_uvector<page_info> d_pages(pages.size(), stream);
    hipMemcpyAsync(d_pages.data(), pages.data(), sizeof(page_info) * pages.size(), hipMemcpyDefault, stream);

    rmm::device_uvector<page_info> cum_pages(page_keys.size(), stream);
    auto page_input = thrust::make_transform_iterator(page_index.begin(), page_info_by_index{ d_pages.data() });
    thrust::inclusive_scan_by_key(thrust::device,
      page_keys.begin(),
      page_keys.end(),
      page_input,
      cum_pages.begin(),
      thrust::equal_to{},
      [] __device__(auto const &a, auto const &b) {
          return page_info{ b.file_idx,
              b.rg_idx,
              b.col_idx,
              b.page_idx,
              b.schema_idx,
              b.global_col_idx,
              a.row_count + b.row_count,
              a.size_bytes + b.size_bytes };
      });

    rmm::device_uvector<page_info> cum_pages_sorted{ cum_pages, stream };
    thrust::sort(thrust::device,
      cum_pages_sorted.begin(),
      cum_pages_sorted.end(),
      [] __device__(page_info const &a, page_info const &b) { return a.row_count < b.row_count; });

    rmm::device_uvector<int> key_offsets(page_keys.size() + 1, stream);
    auto const key_offsets_end = thrust::reduce_by_key(thrust::device,
      page_keys.begin(),
      page_keys.end(),
      thrust::make_constant_iterator(1),
      thrust::make_discard_iterator(),
      key_offsets.begin())
                                   .second;
    size_t const num_unique_keys = key_offsets_end - key_offsets.begin();
    thrust::exclusive_scan(thrust::device, key_offsets.begin(), key_offsets.end(), key_offsets.begin());

    rmm::device_uvector<page_info> aggregated_info(cum_pages.size(), stream);
    thrust::transform(thrust::device,
      cum_pages_sorted.begin(),
      cum_pages_sorted.end(),
      aggregated_info.begin(),
      row_total_size{ cum_pages.data(), key_offsets.data(), num_unique_keys });

    rmm::device_uvector<page_info> d_filtered_pages(aggregated_info.size(), stream);
    auto filtered_end = thrust::copy_if(thrust::device,
      aggregated_info.begin(),
      aggregated_info.end(),
      d_filtered_pages.begin(),
      [sort_col] __device__(auto const &pg) { return pg.schema_idx == sort_col; });
    d_filtered_pages.resize(std::distance(d_filtered_pages.begin(), filtered_end), stream);

    // bring filtered_pages to host for last step
    std::vector<page_info> filtered_pages(d_filtered_pages.size());
    hipMemcpyAsync(filtered_pages.data(),
      d_filtered_pages.data(),
      sizeof(page_info) * d_filtered_pages.size(),
      hipMemcpyDefault,
      stream);
    stream.synchronize();

    std::deque<scalar_pair> ranges;
    size_t cur_pos = 0;
    size_t cur_cumulative_size = 0;
    size_t cur_row_count = 0;
    auto start = thrust::make_transform_iterator(
      filtered_pages.begin(), [&](page_info const &i) { return i.size_bytes - cur_cumulative_size; });
    auto end = start + filtered_pages.size();

    auto last_scalar = min_for_type(col_type, conv_type);
    auto max_scalar = max_for_type(col_type, conv_type);

    std::string last_val = "-inf";
    std::string const max_val = "inf";
    while (true) {
        int64_t split_pos = thrust::lower_bound(thrust::seq, start + cur_pos, end, chunk_read_limit) - start;

        // if we're past the end, or if the returned bucket is > than the chunk_read_limit, move
        // back one.
        if (static_cast<size_t>(split_pos) >= filtered_pages.size()
            || (filtered_pages[split_pos].size_bytes - cur_cumulative_size > chunk_read_limit)) {
            split_pos--;
        }

        // best-try. if we can't find something that'll fit, we have to go bigger. we're doing
        // this in a loop because all of the cumulative sizes for all the pages are sorted into
        // one big list. so if we had two columns, both of which had an entry {1000, 10000},
        // that entry would be in the list twice. so we have to iterate until we skip past all
        // of them.  The idea is that we either do this, or we have to call unique() on the
        // input first.
        while (split_pos < (static_cast<int64_t>(filtered_pages.size()) - 1)
               && (split_pos < 0 || filtered_pages[split_pos].row_count == cur_row_count)) {
            split_pos++;
        }

        auto const start_row = cur_row_count;
        cur_row_count = filtered_pages[split_pos].row_count;

        if (cur_row_count == start_row) break;

        cur_pos = split_pos;
        cur_cumulative_size = filtered_pages[split_pos].size_bytes;

        auto const &splt = filtered_pages[split_pos];
        auto const &colidx = indexes_per_file[splt.file_idx][splt.col_idx];
        auto const &min = colidx.min_values[splt.page_idx];
        auto const &end_val = split_pos == filtered_pages.size() - 1 ? max_val : min;

        auto end_sclr = to_scalar(min, col_type, conv_type);

        ranges.emplace_back(scalar_pair{ to_string(last_val, col_type, conv_type),
          last_scalar,
          to_string(end_val, col_type, conv_type),
          split_pos == filtered_pages.size() - 1 ? max_scalar : end_sclr });
        SPDLOG_INFO(
          "adding range {} -> {}", to_string(last_val, col_type, conv_type), to_string(end_val, col_type, conv_type));

        last_val = min;
        last_scalar = end_sclr;
    }

    return ranges;
}